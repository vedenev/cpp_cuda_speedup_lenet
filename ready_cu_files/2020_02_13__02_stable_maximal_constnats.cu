#include "hip/hip_runtime.h"


#include <iostream>

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <string.h>
#include <ctime>




//const unsigned int W_conv1_size_const = 144;
//__device__ __constant__ float W_conv1_const[W_conv1_size_const];
//__constant__ float W_conv1_const[W_conv1_size_const];
const unsigned int constants_size = (144 + 16 + 6400 + 16 + 256 + 2560 + 10);
__constant__ float constants[constants_size];


__global__ void convolutions_relu_constants_weights(int input_offset, float* features_input, int features_input_size_x, int features_input_size_y, int features_input_n_channels,
	float* features_output, int features_output_size_x, int features_output_size_y, int features_output_n_channels,
	int weights_offset, int weights_size_x, int weights_size_y,
	int biases_offset)
{

	const unsigned int index_output_x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int index_output_y = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int index_output_channel = blockIdx.z * blockDim.z + threadIdx.z;



	unsigned int output_1d_index = features_output_size_y * features_output_size_x * index_output_channel +
		features_output_size_x * index_output_y + index_output_x;

	unsigned int weights_step_1 = weights_size_y * weights_size_x;
	unsigned int weights_step_2 = weights_step_1 * features_input_n_channels * index_output_channel;

	unsigned int features_input_step = features_input_size_y * features_input_size_x;

	float output_value = 0.0;

	for (int index_input_channel = 0; index_input_channel < features_input_n_channels; index_input_channel++)
	{
		unsigned int weights_1d_index_offset = weights_step_2 + weights_step_1 * index_input_channel;

		for (int weights_index_y = 0; weights_index_y < weights_size_y; weights_index_y++) {
			for (int weights_index_x = 0; weights_index_x < weights_size_x; weights_index_x++) {
				unsigned int index_input_x = index_output_x + weights_index_x;
				unsigned int index_input_y = index_output_y + weights_index_y;
				unsigned int input_1d_index = input_offset + features_input_step * index_input_channel +
					features_input_size_x * index_input_y + index_input_x;
				unsigned int weights_1d_index = weights_1d_index_offset + weights_size_x * weights_index_y + weights_index_x;
				output_value += features_input[input_1d_index] * constants[weights_offset + weights_1d_index];

			}
		}

	}

	output_value += constants[biases_offset + index_output_channel];


	output_value = fmaxf(output_value, 0.0); // relu

	features_output[output_1d_index] = output_value;

}

__global__ void convolutions_relu_constants_biases(int input_offset, float* features_input, int features_input_size_x, int features_input_size_y, int features_input_n_channels,
	float* features_output, int features_output_size_x, int features_output_size_y, int features_output_n_channels,
	float* weights, int weights_size_x, int weights_size_y,
	int biases_offset)
{

	const unsigned int index_output_x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int index_output_y = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int index_output_channel = blockIdx.z * blockDim.z + threadIdx.z;



	unsigned int output_1d_index = features_output_size_y * features_output_size_x * index_output_channel +
		features_output_size_x * index_output_y + index_output_x;

	unsigned int weights_step_1 = weights_size_y * weights_size_x;
	unsigned int weights_step_2 = weights_step_1 * features_input_n_channels * index_output_channel;

	unsigned int features_input_step = features_input_size_y * features_input_size_x;

	float output_value = 0.0;

	for (int index_input_channel = 0; index_input_channel < features_input_n_channels; index_input_channel++)
	{
		unsigned int weights_1d_index_offset = weights_step_2 + weights_step_1 * index_input_channel;

		for (int weights_index_y = 0; weights_index_y < weights_size_y; weights_index_y++) {
			for (int weights_index_x = 0; weights_index_x < weights_size_x; weights_index_x++) {
				unsigned int index_input_x = index_output_x + weights_index_x;
				unsigned int index_input_y = index_output_y + weights_index_y;
				unsigned int input_1d_index = input_offset + features_input_step * index_input_channel +
					features_input_size_x * index_input_y + index_input_x;
				unsigned int weights_1d_index = weights_1d_index_offset + weights_size_x * weights_index_y + weights_index_x;
				output_value += features_input[input_1d_index] * weights[weights_1d_index];

			}
		}

	}

	output_value += constants[biases_offset + index_output_channel];


	output_value = fmaxf(output_value, 0.0); // relu

	features_output[output_1d_index] = output_value;

}

__global__ void convolutions_constants_weights(int input_offset, float* features_input, int features_input_size_x, int features_input_size_y, int features_input_n_channels,
	float* features_output, int features_output_size_x, int features_output_size_y, int features_output_n_channels,
	int weights_offset, int weights_size_x, int weights_size_y,
	int biases_offset)
{

	const unsigned int index_output_x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int index_output_y = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int index_output_channel = blockIdx.z * blockDim.z + threadIdx.z;



	unsigned int output_1d_index = features_output_size_y * features_output_size_x * index_output_channel +
		features_output_size_x * index_output_y + index_output_x;

	unsigned int weights_step_1 = weights_size_y * weights_size_x;
	unsigned int weights_step_2 = weights_step_1 * features_input_n_channels * index_output_channel;

	unsigned int features_input_step = features_input_size_y * features_input_size_x;

	float output_value = 0.0;

	for (int index_input_channel = 0; index_input_channel < features_input_n_channels; index_input_channel++)
	{
		unsigned int weights_1d_index_offset = weights_step_2 + weights_step_1 * index_input_channel;

		for (int weights_index_y = 0; weights_index_y < weights_size_y; weights_index_y++) {
			for (int weights_index_x = 0; weights_index_x < weights_size_x; weights_index_x++) {
				unsigned int index_input_x = index_output_x + weights_index_x;
				unsigned int index_input_y = index_output_y + weights_index_y;
				unsigned int input_1d_index = input_offset + features_input_step * index_input_channel +
					features_input_size_x * index_input_y + index_input_x;
				unsigned int weights_1d_index = weights_1d_index_offset + weights_size_x * weights_index_y + weights_index_x;
				output_value += features_input[input_1d_index] * constants[weights_offset + weights_1d_index];

			}
		}

	}

	output_value += constants[biases_offset + index_output_channel];
	features_output[output_1d_index] = output_value;

}

__global__ void max_pooling_2x2(float* features_input, int features_input_size_x, int features_input_size_y_x, int features_input_n_channels, 
	float* features_output, int features_output_size_x, int features_output_size_y_x)
{
	const unsigned int index_output_x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int index_output_y = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int index_output_channel = blockIdx.z * blockDim.z + threadIdx.z;

	unsigned int index_input_x = 2 * index_output_x;
	unsigned int index_input_y = 2 * index_output_y;

	unsigned int output_1d_index = features_output_size_y_x * index_output_channel +
		features_output_size_x * index_output_y + index_output_x;

	unsigned int features_input_step = features_input_size_y_x * index_output_channel;
	unsigned int input_1d_index_0_0 = features_input_step +
		features_input_size_x * index_input_y + index_input_x;
	unsigned int input_1d_index_0_1 = input_1d_index_0_0 + 1;
	unsigned int input_1d_index_1_0 = input_1d_index_0_0 + features_input_size_x;
	unsigned int input_1d_index_1_1 = input_1d_index_0_0 + 1 + features_input_size_x;

	float max_0 = fmaxf(features_input[input_1d_index_0_0], features_input[input_1d_index_0_1]);
	float max_1 = fmaxf(features_input[input_1d_index_1_0], features_input[input_1d_index_1_1]);
	features_output[output_1d_index] = fmaxf(max_0, max_1);
}

__global__ void max_pooling_3x3(float* features_input, int features_input_size_x, int features_input_size_y_x, int features_input_n_channels,
	float* features_output, int features_output_size_x, int features_output_size_y_x)
{
	const unsigned int index_output_x = blockIdx.x * blockDim.x + threadIdx.x;
	const unsigned int index_output_y = blockIdx.y * blockDim.y + threadIdx.y;
	const unsigned int index_output_channel = blockIdx.z * blockDim.z + threadIdx.z;

	unsigned int index_input_x = 3 * index_output_x;
	unsigned int index_input_y = 3 * index_output_y;

	unsigned int output_1d_index = features_output_size_y_x * index_output_channel +
		features_output_size_x * index_output_y + index_output_x;

	unsigned int features_input_step = features_input_size_y_x * index_output_channel;
	unsigned int input_1d_index_0_0 = features_input_step +
		features_input_size_x * index_input_y + index_input_x;
	unsigned int input_1d_index_0_1 = input_1d_index_0_0 + 1;
	unsigned int input_1d_index_0_2 = input_1d_index_0_0 + 2;
	unsigned int input_1d_index_1_0 = input_1d_index_0_0 + features_input_size_x;
	unsigned int input_1d_index_1_1 = input_1d_index_1_0 + 1;
	unsigned int input_1d_index_1_2 = input_1d_index_1_0 + 2;
	unsigned int input_1d_index_2_0 = input_1d_index_1_0 + features_input_size_x;
	unsigned int input_1d_index_2_1 = input_1d_index_2_0 + 1;
	unsigned int input_1d_index_2_2 = input_1d_index_2_0 + 2;

	float max_0 = fmaxf(features_input[input_1d_index_0_0], features_input[input_1d_index_0_1]);
	float max_1 = fmaxf(features_input[input_1d_index_0_2], features_input[input_1d_index_1_0]);
	float max_2 = fmaxf(features_input[input_1d_index_1_1], features_input[input_1d_index_1_2]);
	float max_3 = fmaxf(features_input[input_1d_index_2_0], features_input[input_1d_index_2_1]);

	float max_4 = fmaxf(max_0, max_1);
	float max_5 = fmaxf(max_2, max_3);

	float max_6 = fmaxf(max_4, max_5);

	features_output[output_1d_index] = fmaxf(max_6, features_input[input_1d_index_2_2]);
}

void c_stings_concatinate(char* string_1, char* string_2, char** string_result)
{
	*string_result = (char*)malloc(strlen(string_1) + strlen(string_2) + 1);
	strcpy(*string_result, string_1);
	strcat(*string_result, string_2);

}

void load_data_to_array(char* dir, char* file, float** array_gpu, int size)
{
	// https://stackoverflow.com/questions/22826380/cuda-allocation-and-return-array-from-gpu-to-cpu
	char* path;
	c_stings_concatinate(dir, file, &path);
	float* array_cpu;
	array_cpu = (float*)malloc(sizeof(float) * size);
	FILE* file_id;
	file_id = fopen(path, "rb");
	int n_floats_readed = fread(array_cpu, sizeof(float), size, file_id);
	fclose(file_id);
	if (n_floats_readed != size)
	{
		printf("n_floats_readed != size   n_floats_readed = %d  size = %d\n", n_floats_readed, size);
	}
	if (hipMalloc((void**)array_gpu, sizeof(float) * size) != hipSuccess)
	{
		std::cout << "Error allocating GPU\n";
	}
	
	hipMemcpy(*array_gpu, array_cpu, sizeof(float) * size, hipMemcpyHostToDevice);

	
	//float* array_cpu_check;
	//array_cpu_check = (float*)malloc(sizeof(float) * size);
	//hipMemcpy(array_cpu_check, array_gpu, sizeof(float) * size, hipMemcpyDeviceToHost);
	//for (int counter = size-1; counter < size; counter++)
	//{
	//	printf("array_cpu_check[counter] = %.6f    array_cpu[counter] = %.6f\n", array_cpu_check[counter], array_cpu[counter]);
	//}
	//free(array_cpu_check);
	

	free(array_cpu);
	free(path);
}

void append_data_to_cpu_array(char* dir, char* file, float* constants_cpu, int size, int* offset)
{
	char* path;
	c_stings_concatinate(dir, file, &path);
	float* pointer_shifted;
	FILE* file_id;
	file_id = fopen(path, "rb");
	pointer_shifted = constants_cpu + *offset;
	int n_floats_readed = fread(pointer_shifted, sizeof(float), size, file_id);
	fclose(file_id);
	*offset += size;
	free(path);

}

/*
__global__ void tmp_check(float* array_cpu_4, float* accuracy_ptr)
{
	accuracy_ptr[0] = 0.0;
	for (int weights_1d_index = 0; weights_1d_index < W_conv1_size_const; weights_1d_index++)
	{
		if (W_conv1_const[weights_1d_index] == array_cpu_4[weights_1d_index])
		//if (W_conv1_const[weights_1d_index] == 0.0)
		{
			accuracy_ptr[0] += 1.0;
		}
		//accuracy_ptr[0] += W_conv1_const[weights_1d_index];
	}
	accuracy_ptr[0] /= W_conv1_size_const;
}
*/

__global__ void check_constants(float* array_cpu, int size, float* n_correct_ptr)
{
	for (int index = 0; index < size; index++)
	{
		if (constants[index] == array_cpu[index])
		{
			n_correct_ptr[0] += 1.0;
		}
	}
}

int main(void)
{
	
	
	char* weights_dir = "F:/freelance/cpp_learning/cuda_learning/weigths_1d/";
	



	

	//              c 3 x 3            p 2 x 2           c 5 x 5         p 3 x 3         c 3 x 3          c 1 x 1
	// 28 x 28 x 1   ->   26 x 26 x 16   ->  13 x 13 x 16  ->  9 x 9 x 16 ->   3 x 3 x 16  ->  1 x 1 x 256  ->   1 x 1 x 10
	// n mult : 97344                                 518400                          36864             2560
	// 784                   36864           2704               1296              144                256            10
	int input_size_x = 28;
	int input_size_y = 28;
	int input_n_channels = 1;
	int n_output = 10;

	int featuremaps_1_size_x = 26;
	int featuremaps_1_size_y = 26;
	int featuremaps_1_size_y_x = featuremaps_1_size_x * featuremaps_1_size_y;
	int featuremaps_1_n_channels = 16;
	int featuremaps_1_size = featuremaps_1_size_x * featuremaps_1_size_y * featuremaps_1_n_channels;
	int featuremaps_1_thread_size_x = 13;
	int featuremaps_1_thread_size_y = 13;
	int featuremaps_1_thread_size_z = 4;
	int featuremaps_1_greed_size_x = featuremaps_1_size_x / featuremaps_1_thread_size_x;
	int featuremaps_1_greed_size_y = featuremaps_1_size_y / featuremaps_1_thread_size_y;
	int featuremaps_1_greed_size_z = featuremaps_1_n_channels / featuremaps_1_thread_size_z;

	int featuremaps_1_pooling_size_x = 13;
	int featuremaps_1_pooling_size_y = 13;
	int featuremaps_1_pooling_size_y_x = featuremaps_1_pooling_size_x * featuremaps_1_pooling_size_y;
	int featuremaps_1_pooling_size = featuremaps_1_pooling_size_x * featuremaps_1_pooling_size_y * featuremaps_1_n_channels;
	int featuremaps_1_pooling_thread_size_x = 13;
	int featuremaps_1_pooling_thread_size_y = 13;
	int featuremaps_1_pooling_thread_size_z = 4;
	int featuremaps_1_pooling_greed_size_x = featuremaps_1_pooling_size_x / featuremaps_1_pooling_thread_size_x;
	int featuremaps_1_pooling_greed_size_y = featuremaps_1_pooling_size_y / featuremaps_1_pooling_thread_size_y;
	int featuremaps_1_pooling_greed_size_z = featuremaps_1_n_channels / featuremaps_1_pooling_thread_size_z;

	int featuremaps_2_size_x = 9;
	int featuremaps_2_size_y = 9;
	int featuremaps_2_size_y_x = featuremaps_2_size_x * featuremaps_2_size_y;
	int featuremaps_2_n_channels = 16;
	int featuremaps_2_size = featuremaps_2_size_x * featuremaps_2_size_y * featuremaps_2_n_channels;
	int featuremaps_2_thread_size_x = 3;
	int featuremaps_2_thread_size_y = 3;
	int featuremaps_2_thread_size_z = 4;
	int featuremaps_2_greed_size_x = featuremaps_2_size_x / featuremaps_2_thread_size_x;
	int featuremaps_2_greed_size_y = featuremaps_2_size_y / featuremaps_2_thread_size_y;
	int featuremaps_2_greed_size_z = featuremaps_2_n_channels / featuremaps_2_thread_size_z;

	int featuremaps_2_pooling_size_x = 3;
	int featuremaps_2_pooling_size_y = 3;
	int featuremaps_2_pooling_size_y_x = featuremaps_2_pooling_size_x * featuremaps_2_pooling_size_y;
	int featuremaps_2_pooling_size = featuremaps_2_pooling_size_x * featuremaps_2_pooling_size_y * featuremaps_2_n_channels;
	int featuremaps_2_pooling_thread_size_x = 3;
	int featuremaps_2_pooling_thread_size_y = 3;
	int featuremaps_2_pooling_thread_size_z = 4;
	int featuremaps_2_pooling_greed_size_x = featuremaps_2_pooling_size_x / featuremaps_2_pooling_thread_size_x;
	int featuremaps_2_pooling_greed_size_y = featuremaps_2_pooling_size_y / featuremaps_2_pooling_thread_size_y;
	int featuremaps_2_pooling_greed_size_z = featuremaps_2_n_channels / featuremaps_2_pooling_thread_size_z;

	int featuremaps_3_size_x = 1;
	int featuremaps_3_size_y = 1;
	int featuremaps_3_n_channels = 256;
	int featuremaps_3_size = featuremaps_3_size_x * featuremaps_3_size_y * featuremaps_3_n_channels;
	int featuremaps_3_thread_size_x = 1;
	int featuremaps_3_thread_size_y = 1;
	int featuremaps_3_thread_size_z = 64;
	int featuremaps_3_greed_size_x = featuremaps_3_size_x / featuremaps_3_thread_size_x;
	int featuremaps_3_greed_size_y = featuremaps_3_size_y / featuremaps_3_thread_size_y;
	int featuremaps_3_greed_size_z = featuremaps_3_n_channels / featuremaps_3_thread_size_z;

	int featuremaps_4_size_x = 1;
	int featuremaps_4_size_y = 1;
	int featuremaps_4_n_channels = n_output;
	int featuremaps_4_size = featuremaps_3_size_x * featuremaps_3_size_y * featuremaps_3_n_channels;
	int featuremaps_4_thread_size_x = 1;
	int featuremaps_4_thread_size_y = 1;
	int featuremaps_4_thread_size_z = 10;
	int featuremaps_4_greed_size_x = featuremaps_4_size_x / featuremaps_4_thread_size_x;
	int featuremaps_4_greed_size_y = featuremaps_4_size_y / featuremaps_4_thread_size_y;
	int featuremaps_4_greed_size_z = featuremaps_4_n_channels / featuremaps_4_thread_size_z;

	int W_conv1_size_x = 3;
	int W_conv1_size_y = 3;
	int W_conv1_size = W_conv1_size_x * W_conv1_size_y * input_n_channels * featuremaps_1_n_channels;
	int b_conv1_size = featuremaps_1_n_channels;

	int W_conv2_size_x = 5;
	int W_conv2_size_y = 5;
	int W_conv2_size = W_conv2_size_x * W_conv2_size_y * featuremaps_1_n_channels * featuremaps_2_n_channels;
	int b_conv2_size = featuremaps_2_n_channels;

	int W_conv3_size_x = 3;
	int W_conv3_size_y = 3;
	int W_conv3_size = W_conv3_size_x * W_conv3_size_y * featuremaps_2_n_channels * featuremaps_3_n_channels;
	int b_conv3_size = featuremaps_3_n_channels;

	int W_conv4_size_x = 1;
	int W_conv4_size_y = 1;
	int W_conv4_size = W_conv4_size_x * W_conv4_size_y * featuremaps_3_n_channels * featuremaps_4_n_channels;
	int b_conv4_size = 10;

	int x_val_size = 7840000;
	int n_samples = 10000;
	
	//constants

	float* constants_cpu;
	constants_cpu = (float*)malloc(sizeof(float) * constants_size);
	int offset = 0;
	int offset_W_conv1 = offset;
	append_data_to_cpu_array(weights_dir, "W_conv1.bin", constants_cpu, W_conv1_size, &offset);
	int offset_b_conv1 = offset;
	append_data_to_cpu_array(weights_dir, "b_conv1.bin", constants_cpu, b_conv1_size, &offset);
	int offset_W_conv2 = offset;
	append_data_to_cpu_array(weights_dir, "W_conv2.bin", constants_cpu, W_conv2_size, &offset);
	int offset_b_conv2 = offset;
	append_data_to_cpu_array(weights_dir, "b_conv2.bin", constants_cpu, b_conv2_size, &offset);
	int offset_b_conv3 = offset;
	append_data_to_cpu_array(weights_dir, "b_conv3.bin", constants_cpu, b_conv3_size, &offset);
	int offset_W_conv4 = offset;
	append_data_to_cpu_array(weights_dir, "W_conv4.bin", constants_cpu, W_conv4_size, &offset);
	int offset_b_conv4 = offset;
	append_data_to_cpu_array(weights_dir, "b_conv4.bin", constants_cpu, b_conv4_size, &offset);
	

	//for (int index = 0; index < constants_size; index++)
	//{
	//	printf("%.6f\n", constants_cpu[index]);
	//	
	//}

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(constants), constants_cpu, sizeof(float)* constants_size));
	float* n_correct_ptr;
	if (hipMalloc((void**)&n_correct_ptr, sizeof(float) * 1) != hipSuccess)
	{
		std::cout << "Error allocating GPU n_correct_ptr\n";
	}
	//check_constants<<<1, 1>>>(constants_cpu, constants_size, n_correct_ptr);
	//hipDeviceSynchronize();
	float* n_correct_ptr_cpu;
	n_correct_ptr_cpu = (float*)malloc(sizeof(float) * 1);
	hipMemcpy(n_correct_ptr_cpu, n_correct_ptr, sizeof(float) * 1, hipMemcpyDeviceToHost);
	printf("\n");
	printf("check constants:\n");
	printf("n_correct_ptr_cpu[0] = %.6f\n", n_correct_ptr_cpu[0]);
	printf("constants_size =%d\n", constants_size);
	printf("\n");
	float* constants_cpu_2;
	constants_cpu_2 = (float*)malloc(sizeof(float) * constants_size);
	checkCudaErrors(hipMemcpyFromSymbol(constants_cpu_2, HIP_SYMBOL(constants), sizeof(float) * constants_size));
	int is_equal = 0;
	int last_correct_index = 0;
	for (int index = 0; index < constants_size; index++)
	{
		if (constants_cpu_2[index] == constants_cpu[index])
		{
			is_equal = 1;
			last_correct_index = index;
		}
		else
		{
			is_equal = 0;
		}
		printf("%.6f   %.6f   %d\n", constants_cpu_2[index], constants_cpu[index], is_equal);
	}
	printf("last_correct_index = %d\n", last_correct_index);
	//last_correct_index = 6792
	hipFree(n_correct_ptr);
	free(n_correct_ptr_cpu);
	


	float* W_conv3;
	float* x_val;
	
	//W_conv1_1d.size = 144
	//b_conv1_1d.size = 16
	//W_conv2_1d.size = 6400
	//b_conv2_1d.size = 16
	//W_conv3_1d.size = 36864
	//b_conv3_1d.size = 256
	//W_conv4_1d.size = 2560
	//b_conv4_1d.size = 10
	//x_val_1d.size = 7840000
	//y_val.size = 10000

	//(144 + 16 + 6400 + 16    +   256 + 2560 + 10)*4 = 37608
	
	load_data_to_array(weights_dir, "W_conv3.bin", &W_conv3, W_conv3_size);
	load_data_to_array(weights_dir, "x_val.bin", &x_val, x_val_size);
	
	char* y_val_cpu;
	y_val_cpu = (char*)malloc(sizeof(char) * n_samples);
	FILE* file_id;
	char* path;
	c_stings_concatinate(weights_dir, "y_val.bin", &path);
	file_id = fopen(path, "rb");
	fread(y_val_cpu, sizeof(float), n_samples, file_id);
	fclose(file_id);
	free(path);

	float* featuremaps_1;
	if (hipMalloc((void**)&featuremaps_1, sizeof(float) * featuremaps_1_size) != hipSuccess)
	{
		std::cout << "Error allocating GPU featuremaps_1\n";
	}

	float* featuremaps_1_pooling;
	if (hipMalloc((void**)&featuremaps_1_pooling, sizeof(float) * featuremaps_1_pooling_size) != hipSuccess)
	{
		std::cout << "Error allocating GPU featuremaps_1_pooling\n";
	}

	float* featuremaps_2;
	if (hipMalloc((void**)&featuremaps_2, sizeof(float) * featuremaps_2_size) != hipSuccess)
	{
		std::cout << "Error allocating GPU featuremaps_2\n";
	}

	float* featuremaps_2_pooling;
	if (hipMalloc((void**)&featuremaps_2_pooling, sizeof(float) * featuremaps_2_pooling_size) != hipSuccess)
	{
		std::cout << "Error allocating GPU featuremaps_2_pooling\n";
	}

	float* featuremaps_3;
	if (hipMalloc((void**)&featuremaps_3, sizeof(float) * featuremaps_3_size) != hipSuccess)
	{
		std::cout << "Error allocating GPU featuremaps_3\n";
	}

	float* featuremaps_4;
	if (hipMalloc((void**)&featuremaps_4, sizeof(float) * featuremaps_4_size) != hipSuccess)
	{
		std::cout << "Error allocating GPU featuremaps_4\n";
	}

	int input_step = input_size_x * input_size_y * input_n_channels;

	dim3 grid_featuremaps_1(featuremaps_1_greed_size_x, featuremaps_1_greed_size_y, featuremaps_1_greed_size_z);
	dim3 threadBlock_featuremaps_1(featuremaps_1_thread_size_x, featuremaps_1_thread_size_y, featuremaps_1_thread_size_z);

	dim3 grid_featuremaps_1_pooling(featuremaps_1_pooling_greed_size_x, featuremaps_1_pooling_greed_size_y, featuremaps_1_pooling_greed_size_z);
	dim3 threadBlock_featuremaps_1_pooling(featuremaps_1_pooling_thread_size_x, featuremaps_1_pooling_thread_size_y, featuremaps_1_pooling_thread_size_z);

	dim3 grid_featuremaps_2(featuremaps_2_greed_size_x, featuremaps_2_greed_size_y, featuremaps_2_greed_size_z);
	dim3 threadBlock_featuremaps_2(featuremaps_2_thread_size_x, featuremaps_2_thread_size_y, featuremaps_2_thread_size_z);

	dim3 grid_featuremaps_2_pooling(featuremaps_2_pooling_greed_size_x, featuremaps_2_pooling_greed_size_y, featuremaps_2_pooling_greed_size_z);
	dim3 threadBlock_featuremaps_2_pooling(featuremaps_2_pooling_thread_size_x, featuremaps_2_pooling_thread_size_y, featuremaps_2_pooling_thread_size_z);

	dim3 grid_featuremaps_3(featuremaps_3_greed_size_x, featuremaps_3_greed_size_y, featuremaps_3_greed_size_z);
	dim3 threadBlock_featuremaps_3(featuremaps_3_thread_size_x, featuremaps_3_thread_size_y, featuremaps_3_thread_size_z);

	dim3 grid_featuremaps_4(featuremaps_4_greed_size_x, featuremaps_4_greed_size_y, featuremaps_4_greed_size_z);
	dim3 threadBlock_featuremaps_4(featuremaps_4_thread_size_x, featuremaps_4_thread_size_y, featuremaps_4_thread_size_z);


	//dim3 grid_featuremaps_1(2, 2, 4);
	//dim3 threadBlock_featuremaps_1(13, 13, 4);

	//printf("featuremaps_1_size = %d\n", featuremaps_1_size);
	//printf("sizeof(featuremaps_1) = %d\n", sizeof(featuremaps_1));
	//printf("sizeof(W_conv1) = %d\n", sizeof(W_conv1));
	//printf("sizeof(b_conv1) = %d\n", sizeof(b_conv1));
	//size_t b_conv1_sized = 0;
	//hipError_t er1 = hipGetSymbolSize(&b_conv1_sized, HIP_SYMBOL(b_conv1));
	//printf("b_conv1_sized = %d\n", b_conv1_sized);
	//size_t featuremaps_1_sized = 0;
	//hipError_t er2 = hipGetSymbolSize(&featuremaps_1_sized, HIP_SYMBOL(featuremaps_1));
	//printf("featuremaps_1_sized = %d\n", featuremaps_1_sized);

	//dim3 grid_featuremaps_1(1, 1, 1);
	//dim3 threadBlock_featuremaps_1(1, 1, 1);


	
	//convolutions_relu<<<grid_featuremaps_1, threadBlock_featuremaps_1>>>(0, x_val, input_size_x, input_size_y, input_n_channels,
	//	featuremaps_1, featuremaps_1_size_x, featuremaps_1_size_y, featuremaps_1_n_channels,
	//	W_conv1, W_conv1_size_x, W_conv2_size_x,
	//	b_conv1);
	//hipDeviceSynchronize();
	
	

	float* featuremaps_4_tmp_cpu;
	featuremaps_4_tmp_cpu = (float*)malloc(sizeof(float) * featuremaps_4_size);
	float featuremaps_4_max = 0.0;
	int featuremaps_4_max_ind = -1;
	int n_correct_answers = 0;
	clock_t begin = clock();
	for (int sample_count = 0; sample_count < n_samples; sample_count++)
	{
		
		int input_offset = sample_count * input_step;
		convolutions_relu_constants_weights<<<grid_featuremaps_1, threadBlock_featuremaps_1>>>(input_offset, x_val, input_size_x, input_size_y, input_n_channels,
			featuremaps_1, featuremaps_1_size_x, featuremaps_1_size_y, featuremaps_1_n_channels,
			offset_W_conv1, W_conv1_size_x, W_conv1_size_y,
			offset_b_conv1);

		
		hipDeviceSynchronize();

		
		max_pooling_2x2<<<grid_featuremaps_1_pooling, threadBlock_featuremaps_1_pooling>>> (featuremaps_1, featuremaps_1_size_x, featuremaps_1_size_y_x, featuremaps_1_n_channels,
			featuremaps_1_pooling, featuremaps_1_pooling_size_x, featuremaps_1_pooling_size_y_x);

		hipDeviceSynchronize();

		convolutions_relu_constants_weights<<<grid_featuremaps_2, threadBlock_featuremaps_2>>>(0, featuremaps_1_pooling, featuremaps_1_pooling_size_x, featuremaps_1_pooling_size_y, featuremaps_1_n_channels,
			featuremaps_2, featuremaps_2_size_x, featuremaps_2_size_y, featuremaps_2_n_channels,
			offset_W_conv2, W_conv2_size_x, W_conv2_size_y,
			offset_b_conv2);

		hipDeviceSynchronize();

		max_pooling_3x3<<<grid_featuremaps_2_pooling, threadBlock_featuremaps_2_pooling>>>(featuremaps_2, featuremaps_2_size_x, featuremaps_2_size_y_x, featuremaps_2_n_channels,
			featuremaps_2_pooling, featuremaps_2_pooling_size_x, featuremaps_2_pooling_size_y_x);

		hipDeviceSynchronize();

		convolutions_relu_constants_biases<<<grid_featuremaps_3, threadBlock_featuremaps_3>>> (0, featuremaps_2_pooling, featuremaps_2_pooling_size_x, featuremaps_2_pooling_size_y, featuremaps_2_n_channels,
			featuremaps_3, featuremaps_3_size_x, featuremaps_3_size_y, featuremaps_3_n_channels,
			W_conv3, W_conv3_size_x, W_conv3_size_y,
			offset_b_conv3);

		hipDeviceSynchronize();

		convolutions_constants_weights<<<grid_featuremaps_4, threadBlock_featuremaps_4>>>(0, featuremaps_3, featuremaps_3_size_x, featuremaps_3_size_y, featuremaps_3_n_channels,
			featuremaps_4, featuremaps_4_size_x, featuremaps_4_size_y, featuremaps_4_n_channels,
			offset_W_conv4, W_conv4_size_x, W_conv4_size_y,
			offset_b_conv4);

		hipDeviceSynchronize();

		hipMemcpy(featuremaps_4_tmp_cpu, featuremaps_4, sizeof(float)* featuremaps_4_size, hipMemcpyDeviceToHost);

		
		featuremaps_4_max = featuremaps_4_tmp_cpu[0];
		featuremaps_4_max_ind = 0;
		for (int output_index = 1; output_index < n_output; output_index++)
		{
			//printf("output_index = %d\n", output_index);
			if (featuremaps_4_tmp_cpu[output_index] > featuremaps_4_max)
			{
				featuremaps_4_max = featuremaps_4_tmp_cpu[output_index];
				featuremaps_4_max_ind = output_index;
				//printf("featuremaps_4_max = %.6fd\n", featuremaps_4_max);
				//printf("featuremaps_4_max_ind = %d\n", featuremaps_4_max_ind);
			}
		}
		//printf("featuremaps_4_max_ind =%d\n", featuremaps_4_max_ind);
		//printf("y_val_cpu[sample_count] =%d\n", y_val_cpu[sample_count]);
		if (featuremaps_4_max_ind == y_val_cpu[sample_count])
		{
			n_correct_answers++;
		}

		

		
			
	}
	clock_t end = clock();
	double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
	double time_mean = elapsed_secs / n_samples;

	float accuracy = ((float)n_correct_answers) / n_samples;
	printf("accuracy = %.8f\n", accuracy);
	printf("elapsed_secs = %.8f\n", elapsed_secs);
	printf("time_mean = %.8f\n", time_mean);
	
	

	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "ERROR: %s\n", hipGetErrorString(error));
	}

	
	
	
	//float* featuremaps_1_tmp_cpu;
	//featuremaps_1_tmp_cpu = (float*)malloc(sizeof(float) * featuremaps_1_size);
	//hipMemcpy(featuremaps_1_tmp_cpu, featuremaps_1, sizeof(float) * featuremaps_1_size, hipMemcpyDeviceToHost);
	//printf("featuremaps_1_tmp_cpu[0] = %.6f\n", featuremaps_1_tmp_cpu[0]);
	//printf("featuremaps_1_tmp_cpu[1] = %.6f\n", featuremaps_1_tmp_cpu[1]);
	//printf("featuremaps_1_tmp_cpu[2] = %.6f\n", featuremaps_1_tmp_cpu[2]);
	//printf("featuremaps_1_tmp_cpu[20] = %.6f\n", featuremaps_1_tmp_cpu[20]);
	//printf("featuremaps_1_tmp_cpu[200] = %.6f\n", featuremaps_1_tmp_cpu[200]);
	//printf("featuremaps_1_tmp_cpu[1000] = %.6f\n", featuremaps_1_tmp_cpu[1000]);
	//printf("featuremaps_1_tmp_cpu[2000] = %.6f\n", featuremaps_1_tmp_cpu[2000]);
	//printf("featuremaps_1_tmp_cpu[3000] = %.6f\n", featuremaps_1_tmp_cpu[3000]);
	//free(featuremaps_1_tmp_cpu);
	

	
	//float* featuremaps_1_pooling_tmp_cpu;
	//featuremaps_1_pooling_tmp_cpu = (float*)malloc(sizeof(float) * featuremaps_1_pooling_size);
	//hipMemcpy(featuremaps_1_pooling_tmp_cpu, featuremaps_1_pooling, sizeof(float) * featuremaps_1_pooling_size, hipMemcpyDeviceToHost);
	//printf("featuremaps_1_pooling_tmp_cpu[0] = %.6f\n", featuremaps_1_pooling_tmp_cpu[0]);
	//printf("featuremaps_1_pooling_tmp_cpu[1] = %.6f\n", featuremaps_1_pooling_tmp_cpu[1]);
	//printf("featuremaps_1_pooling_tmp_cpu[2] = %.6f\n", featuremaps_1_pooling_tmp_cpu[2]);
	//printf("featuremaps_1_pooling_tmp_cpu[20] = %.6f\n", featuremaps_1_pooling_tmp_cpu[20]);
	//printf("featuremaps_1_pooling_tmp_cpu[200] = %.6f\n", featuremaps_1_pooling_tmp_cpu[200]);
	//printf("featuremaps_1_pooling_tmp_cpu[1000] = %.6f\n", featuremaps_1_pooling_tmp_cpu[1000]);
	//printf("featuremaps_1_pooling_tmp_cpu[2000] = %.6f\n", featuremaps_1_pooling_tmp_cpu[2000]);
	//free(featuremaps_1_pooling_tmp_cpu);
	

	
	//float* featuremaps_2_tmp_cpu;
	//featuremaps_2_tmp_cpu = (float*)malloc(sizeof(float) * featuremaps_2_size);
	//hipMemcpy(featuremaps_2_tmp_cpu, featuremaps_2, sizeof(float) * featuremaps_2_size, hipMemcpyDeviceToHost);
	//printf("featuremaps_2_tmp_cpu[0] = %.6f\n", featuremaps_2_tmp_cpu[0]);
	//printf("featuremaps_2_tmp_cpu[1] = %.6f\n", featuremaps_2_tmp_cpu[1]);
	//printf("featuremaps_2_tmp_cpu[2] = %.6f\n", featuremaps_2_tmp_cpu[2]);
	//printf("featuremaps_2_tmp_cpu[20] = %.6f\n", featuremaps_2_tmp_cpu[20]);
	//printf("featuremaps_2_tmp_cpu[200] = %.6f\n", featuremaps_2_tmp_cpu[200]);
	//printf("featuremaps_2_tmp_cpu[1000] = %.6f\n", featuremaps_2_tmp_cpu[1000]);
	//free(featuremaps_2_tmp_cpu);
	

	
	
	//printf("featuremaps_4_tmp_cpu[0] = %.6f\n", featuremaps_4_tmp_cpu[0]);
	//printf("featuremaps_4_tmp_cpu[1] = %.6f\n", featuremaps_4_tmp_cpu[1]);
	//printf("featuremaps_4_tmp_cpu[2] = %.6f\n", featuremaps_4_tmp_cpu[2]);
	//printf("featuremaps_4_tmp_cpu[3] = %.6f\n", featuremaps_4_tmp_cpu[3]);
	//printf("featuremaps_4_tmp_cpu[4] = %.6f\n", featuremaps_4_tmp_cpu[4]);
	//printf("featuremaps_4_tmp_cpu[5] = %.6f\n", featuremaps_4_tmp_cpu[5]);
	//printf("featuremaps_4_tmp_cpu[6] = %.6f\n", featuremaps_4_tmp_cpu[6]);
	//printf("featuremaps_4_tmp_cpu[7] = %.6f\n", featuremaps_4_tmp_cpu[7]);
	//printf("featuremaps_4_tmp_cpu[8] = %.6f\n", featuremaps_4_tmp_cpu[8]);
	//printf("featuremaps_4_tmp_cpu[9] = %.6f\n", featuremaps_4_tmp_cpu[9]);
	
	
	free(featuremaps_4_tmp_cpu);
	free(y_val_cpu);

	
	
	hipFree(x_val);
	hipFree(featuremaps_1);
	hipFree(featuremaps_1_pooling);
	hipFree(featuremaps_2);
	hipFree(featuremaps_2_pooling);
	hipFree(featuremaps_3);
	hipFree(featuremaps_4);

	
	hipFree(W_conv3);
	
	
	
	
	return 0;


}